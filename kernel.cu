#include "hip/hip_runtime.h"


#include "hip/hip_runtime.h"
#include ""

#include <cstdio>
#include <cstdlib>
#include <cmath>

#include "parameters.h"
#include "util.h"
#include "cuda_osem.cuh"


void loadImage(float *d_image1D, int nBytes, int len, char *fileName);

void loadImagePinnedMem(float *h_data, float *d_data, int nBytes, int len, char *fileName);

void saveImage(float *d_data1D, int len, int nBytes, char *fileName);

inline int iDivUp(int a, int b)
{
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

inline float getAngle(int view)
{
	return S0 + view * SLEN / 360.0f;
}

inline void startGPUTimer(hipEvent_t start)
{
	HANDLE_ERROR(hipEventRecord(start));
}

inline void stopGPUTimer(hipEvent_t stop)
{
	HANDLE_ERROR(hipEventRecord(stop));
	HANDLE_ERROR(hipEventSynchronize(stop));
}

inline float getElapsedTime(hipEvent_t start, hipEvent_t stop)
{
	float ms;
	HANDLE_ERROR(hipEventElapsedTime(&ms, start, stop));
	return ms;
}

int main()
{
	//GPU timer
	hipEvent_t start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	float ms;
	
	//1D block for general use
	dim3 block(512);
	dim3 grid(0);

	//2D block for detector array (forward projection)
	dim3 block2d(32, 16);
	dim3 grid2d(iDivUp(NU, block2d.x), iDivUp(NV, block2d.y));

	//3D block for image volume (back projection)
	dim3 block3d(16, 8, 8);
	dim3 grid3d(iDivUp(NX, block3d.x), iDivUp(NY, block3d.y), iDivUp(NZ, block3d.z));

	//All memory allocation
	char *fileName = (char *)malloc(sizeof(char) * 512);

	float *h_data, *data, *proj, *image, *diffImage, *normImage;

	HANDLE_ERROR(hipMalloc((void **)&data, PROJ_BYTES));
	HANDLE_ERROR(hipMalloc((void **)&proj, PROJ_BYTES));
	HANDLE_ERROR(hipMalloc((void **)&image, IMAGE_BYTES));
	HANDLE_ERROR(hipMalloc((void **)&diffImage, IMAGE_BYTES));
	HANDLE_ERROR(hipMalloc((void **)&normImage, IMAGE_BYTES));
	HANDLE_ERROR(hipHostMalloc((void **)&h_data, PROJ_BYTES));

	//CUDA array for 3D texture
	const hipExtent volumeSize = make_hipExtent(NX, NY, NZ);
	createCudaArray(volumeSize);

	float angle, cosbeta, sinbeta;
	int deltaBeta = 1;

	float centernBins = (NU - 1.0) / 2.0;
	float ScaleFactor = R / D;
	float deltaS = DU * ScaleFactor;
	float centernZZ = (NV - 1.0) / 2.0;
	float deltaZZ0 = DV;
	float deltaZZ = deltaZZ0 * ScaleFactor;

	float lamda = LAMDA;

	//reconstruction
	printf("\n --- STARTING RECONSTRUCTION WITH OSEM ---\n\n");

	getGridDim(&grid, block, IMAGE_LEN);
	setArrVal <<<grid, block>>> (image, IMAGE_LEN, 1);
	HANDLE_ERROR(hipMemset(diffImage, 0, IMAGE_BYTES));

	for (int i = 0; i < ITER; i++)
	{
		printf("Iteration : %03d ... ", i + 1);
		startGPUTimer(start);

		bind3DTexture1(image, volumeSize);
		HANDLE_ERROR(hipMemset(normImage, 0, IMAGE_BYTES));

		for (int j = i % N_SUBSET; j < NS; j += N_SUBSET)
		{
			angle = getAngle(j);
			cosbeta = cosf(angle);
			sinbeta = sinf(angle);

			generateFileName(fileName, INPUT_DIR, "", j, ".dat");
			loadImagePinnedMem(h_data, data, PROJ_BYTES, PROJ_LEN, fileName);

			forwardProjectionTexKernel <<<grid2d, block2d>>> (proj, sinbeta, cosbeta,
				R, D, NU, NV, DU, DV, U0, V0, DX, DY, DZ, X0, Y0, Z0, NX, NY, NZ, XLEN, YLEN, ZLEN);

			getGridDim(&grid, block, PROJ_LEN);
			divisionKernel <<<grid, block>>> (data, proj, PROJ_LEN);

			backProjectionOSEMKernel <<<grid3d, block3d>>> (diffImage, data, R, cosbeta,
				sinbeta, deltaZZ, centernZZ, deltaS, centernBins, NV, NU, deltaBeta, normImage,
				NX, NY, NZ, DX, DY, DZ);

			getGridDim(&grid, block, IMAGE_LEN);
			nanAndInfCheck <<<grid, block>>> (diffImage, IMAGE_LEN);
		}

		OSEMUpdateKernel <<<grid, block>>> (image, diffImage, normImage, IMAGE_LEN, lamda);
		makePositive <<<grid, block>>> (image, IMAGE_LEN);
		HANDLE_ERROR(hipMemset(diffImage, 0, IMAGE_BYTES));

		lamda *= REG_FAC;

		stopGPUTimer(stop);
		ms = getElapsedTime(start, stop);
		printf("Elapsed time : %.3f ms\n", ms);

		if (i == 0 || (i + 1) % SAVE_INTERVAL == 0)
		{
			generateFileName(fileName, OUTPUT_DIR, SAVE_FILE_NAME, i + 1, ".dat");
			saveImage(image, IMAGE_LEN, IMAGE_BYTES, fileName);
		}
	}

	printf("\n --- RECONSTRUCTION FINISHED AFTER %d ITERATIONS ---\n\n", ITER);

	//Free all allocated memory
	free3DTexture();
	free(fileName);
	HANDLE_ERROR(hipFree(data));
	HANDLE_ERROR(hipFree(proj));
	HANDLE_ERROR(hipFree(image));
	HANDLE_ERROR(hipFree(diffImage));
	HANDLE_ERROR(hipFree(normImage));
	HANDLE_ERROR(hipHostFree(h_data));
	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));

	//Reset device state
	HANDLE_ERROR(hipDeviceReset());

	return 0;
}

void loadImage(float *d_image1D, int nBytes, int len, char *fileName)
{
	float *h_image1D = (float*)malloc(nBytes);
	readArrayFromFile(h_image1D, len, fileName, sizeof(float));
	HANDLE_ERROR(hipMemcpy(d_image1D, h_image1D, nBytes, hipMemcpyHostToDevice));
	free(h_image1D);
}

void loadImagePinnedMem(float *h_data, float *d_data, int nBytes, int len, char *fileName)
{
	readArrayFromFile(h_data, PROJ_LEN, fileName, sizeof(float));
	HANDLE_ERROR(hipMemcpy(d_data, h_data, PROJ_BYTES, hipMemcpyHostToDevice));
}

void saveImage(float *d_data1D, int len, int nBytes, char *fileName)
{
	float *h_data1D = (float*)malloc(nBytes);
	HANDLE_ERROR(hipMemcpy(h_data1D, d_data1D, nBytes, hipMemcpyDeviceToHost));
	writeArrayToFile(h_data1D, len, fileName, sizeof(float));
	free(h_data1D);
	printf("Saved file \"%s\"\n", fileName);
}